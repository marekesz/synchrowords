#include "hip/hip_runtime.h"
#include "preprocessed_transition_kernel.hpp"

#include <external/cuda/hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <synchrolib/data_structures/preprocessed_transition.hpp>
#include <synchrolib/data_structures/subset.hpp>
#include <synchrolib/utils/bits.hpp>
#include <synchrolib/utils/general.hpp>

#include <chrono>
#include <iostream>

#define BLK 256

namespace {

template <synchrolib::uint N, synchrolib::uint K, synchrolib::uint B, synchrolib::uint S>
__global__ void preprocessed_transition_kernel_impl(
    const synchrolib::PreprocessedTransition<N, K>* trans,
    const synchrolib::Subset<N>* from,
    synchrolib::GpuSubset<N>* to,
    size_t cnt) {

  int ind = blockIdx.x * BLK + threadIdx.x;
  if (ind >= cnt) {
    return;
  }

  synchrolib::Subset<N> sub = from[ind];
  synchrolib::GpuSubset<N> ret;
  for (synchrolib::uint i = 0; i < B; ++i) {
    ret.v[i] = 0;
  }

  for (synchrolib::uint i = 0; i < S; i++) {
    const synchrolib::uint b = i * synchrolib::PREPROCESSED_TRANSITION_SLICE / synchrolib::SUBSETS_BITS;
    const synchrolib::uint shift = i * synchrolib::PREPROCESSED_TRANSITION_SLICE % synchrolib::SUBSETS_BITS;
    const synchrolib::uint set = (synchrolib::uint)(
        (sub.v[b] >> (shift)) & ((1 << synchrolib::PREPROCESSED_TRANSITION_SLICE) - 1));
    for (synchrolib::uint b = 0; b < B; ++b) {
      ret.v[b] |= trans->trans[i][set].v[b];
    }
  }

  to[ind] = ret;
}

} // namespace

namespace synchrolib {

template<uint N, uint K>
void preprocessed_transition_kernel(const PreprocessedTransition<N, K>& trans, const Subset<N>* from, Subset<N>* to, size_t cnt) {
  static_assert(sizeof(Subset<N>) == sizeof(GpuSubset<N>));
  // for (size_t i = 0; i < cnt; ++i, ++from, ++to) {
  //   trans.apply(*from, *to);
  // }
  const PreprocessedTransition<N, K>* trans_ptr;
  const Subset<N>* from_ptr;
  GpuSubset<N>* to_ptr;

// std::chrono::steady_clock::time_point begin, end;
// begin = std::chrono::steady_clock::now();
  checkCudaErrors(hipMalloc((void **) &trans_ptr, sizeof(PreprocessedTransition<N, K>)));
  checkCudaErrors(hipMalloc((void **) &from_ptr, sizeof(Subset<N>) * cnt));
  checkCudaErrors(hipMalloc((void **) &to_ptr, sizeof(Subset<N>) * cnt));

  checkCudaErrors(hipMemcpy((void*) trans_ptr, (void*) &trans, sizeof(PreprocessedTransition<N, K>), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy((void*) from_ptr, (void*) from, sizeof(Subset<N>) * cnt, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(to_ptr, 0, sizeof(Subset<N>) * cnt));
// end = std::chrono::steady_clock::now();
// std::cout << "Copy time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;


// begin = std::chrono::steady_clock::now();
  preprocessed_transition_kernel_impl<N, K, Subset<N>::buckets(), PreprocessedTransition<N, K>::slices()><<<(cnt + BLK - 1) / BLK, BLK/*, 0, stream*/>>>(
    trans_ptr, from_ptr, to_ptr, cnt);
  getLastCudaError("preprocessed_transition_kernel run failed\n"); // TODO: better error message

  checkCudaErrors(hipMemcpy(to, to_ptr, sizeof(Subset<N>) * cnt, hipMemcpyDeviceToHost));
// end = std::chrono::steady_clock::now();
// std::cout << "Gpu time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;

  checkCudaErrors(hipFree((void*) trans_ptr));
  checkCudaErrors(hipFree((void*) from_ptr));
  checkCudaErrors(hipFree(to_ptr));
}

template void preprocessed_transition_kernel<AUT_N, AUT_K>(const PreprocessedTransition<AUT_N, AUT_K>& trans, const Subset<AUT_N>* from, Subset<AUT_N>* to, size_t cnt);

}  // namespace synchrolib

#undef BLK
