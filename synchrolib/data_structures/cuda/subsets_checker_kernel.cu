#include "hip/hip_runtime.h"
#include <synchrolib/data_structures/cuda/subsets_checker_kernel.hpp>

#include <external/cuda/hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace {

__global__ void check_subset_intersection_kernel_impl_(synchrolib::uint64* small, synchrolib::uint64* large, bool* results, int small_cnt, int large_cnt, int buckets, int thresh) {
    int ind = threadIdx.x + blockIdx.x * blockDim.x;
    if (ind >= small_cnt) return;

    int my = ind * buckets;

    int cnt = 0;

    for (int i = 0; i < large_cnt; ++i) {
        int they = i * buckets;

        bool good = true;
        for (int j = 0; j < buckets; ++j) {
            synchrolib::uint64 sm = small[my + j];
            if ((sm & large[they + j]) != sm) {
                good = false;
            }
        }

        if (good) {
            cnt++;
        }
    }

    results[ind] = (cnt >= thresh);
}

} // namespace


namespace synchrolib {

std::unique_ptr<bool[]> check_subset_intersection_kernel_impl(const uint64* small, const uint64* large, int small_cnt, int large_cnt, int buckets, int thresh) {
    uint64* small_ptr;
    uint64* large_ptr;
    bool* results;

    checkCudaErrors(hipMalloc((void **) &small_ptr, sizeof(uint64) * buckets * small_cnt));
    checkCudaErrors(hipMalloc((void **) &large_ptr, sizeof(uint64) * buckets * large_cnt));
    checkCudaErrors(hipMalloc((void **) &results, sizeof(bool) * small_cnt));

    checkCudaErrors(hipMemcpy(small_ptr, small, sizeof(uint64) * buckets * small_cnt, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(large_ptr, large, sizeof(uint64) * buckets * large_cnt, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(results, 0, sizeof(bool) * small_cnt));

    constexpr int threads = 1024;
    check_subset_intersection_kernel_impl_<<<(small_cnt + threads - 1) / threads, threads>>>(
        small_ptr, large_ptr, results, small_cnt, large_cnt, buckets, thresh);
    getLastCudaError("check_subset_intersection_kernel run failed\n"); // TODO: better error message

    auto results_array = std::make_unique<bool[]>(small_cnt);
    checkCudaErrors(hipMemcpy(results_array.get(), results, sizeof(bool) * small_cnt, hipMemcpyDeviceToHost));

    return results_array;
}

void reset_cuda_device_impl() {
    hipDeviceReset();
}

} // namespace synchrolib