#include "hip/hip_runtime.h"
#include <synchrolib/data_structures/cuda/subsets_implicit_trie_kernel.hpp>

#include <external/cuda/hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define BLK 256

namespace {

template<synchrolib::uint N, synchrolib::uint B>
__global__ void subsets_trie_kernel_impl_v2(
    synchrolib::Subset<N>* small,
    synchrolib::Subset<N>* large,
    int small_cnt,
    int large_cnt,
    bool* results) {
  int id = blockIdx.x * BLK + threadIdx.x;
  if (id >= large_cnt) {
    return;
  }

  bool ret = false;
  for (int i = 0; i < small_cnt; ++i) {
    synchrolib::Subset<N> they = small[i];

    bool good = true;
    for (int j = 0; j < B; ++j) {
      if ((large[id].v[j] & they.v[j]) != they.v[j]) {
        good = false;
      }
    }

    if (good) {
      ret = true;
    }
  }

  // if (__any_sync(0xffffffff, ret)) {
  //   if (t == 0) {
  //     results[blockIdx.x] = true;
  //   }
  // }
  if (ret) {
    results[id] = true; // TODO: test speed
  }
}

template<synchrolib::uint N, synchrolib::uint B>
__global__ void subsets_trie_proper_kernel_impl_v2(
    synchrolib::Subset<N>* small,
    synchrolib::Subset<N>* large,
    int small_cnt,
    int large_cnt,
    bool* results) {
  int id = blockIdx.x * BLK + threadIdx.x;
  if (id >= large_cnt) {
    return;
  }

  bool ret = false;
  for (int i = 0; i < small_cnt; ++i) {
    synchrolib::Subset<N> they = small[i];

    bool good = true;
    bool same = true;
    for (int j = 0; j < B; ++j) {
      if ((large[id].v[j] & they.v[j]) != they.v[j]) {
        good = false;
      }

      if (large[id].v[j] != they.v[j]) {
        same = false;
      }
    }

    if (good && !same) {
      ret = true;
    }
  }

  // if (__any_sync(0xffffffff, ret)) {
  //   if (threadIdx.x == 0) {
  //     results[blockIdx.x] = true;
  //   }
  // }
  if (ret) {
    results[id] = true; // TODO: test speed
  }
}




template<synchrolib::uint N, synchrolib::uint B>
__global__ void subsets_trie_kernel_impl(
    synchrolib::Subset<N>* small,
    synchrolib::Subset<N>* large,
    int small_cnt,
    bool* results) {
  synchrolib::Subset<N> sub = large[blockIdx.x];

  bool ret = false;
  for (int i = threadIdx.x; i < small_cnt; i += blockDim.x) {
    synchrolib::Subset<N> they = small[i];

    bool good = true;
    for (int j = 0; j < B; ++j) {
      if ((sub.v[j] & they.v[j]) != they.v[j]) {
        good = false;
      }
    }

    if (good) {
      ret = true;
    }
  }

  // if (__any_sync(0xffffffff, ret)) {
  //   if (threadIdx.x == 0) {
  //     results[blockIdx.x] = true;
  //   }
  // }
  if (ret) {
    results[blockIdx.x] = true; // TODO: test speed
  }
}

template<synchrolib::uint N, synchrolib::uint B>
__global__ void subsets_trie_proper_kernel_impl(
    synchrolib::Subset<N>* small,
    synchrolib::Subset<N>* large,
    int small_cnt,
    bool* results) {
  synchrolib::Subset<N> sub = large[blockIdx.x];

  bool ret = false;
  for (int i = threadIdx.x; i < small_cnt; i += blockDim.x) {
    synchrolib::Subset<N> they = small[i];

    bool good = true;
    bool same = true;
    for (int j = 0; j < B; ++j) {
      if ((sub.v[j] & they.v[j]) != they.v[j]) {
        good = false;
      }

      if (sub.v[j] != they.v[j]) {
        same = false;
      }
    }

    if (good && !same) {
      ret = true;
    }
  }

  // if (__any_sync(0xffffffff, ret)) {
  //   if (threadIdx.x == 0) {
  //     results[blockIdx.x] = true;
  //   }
  // }
  if (ret) {
    results[blockIdx.x] = true; // TODO: test speed
  }
}

} // namespace


namespace synchrolib {

template<uint N, bool Proper>
SubsetsImplicitTrieKernel<N, Proper>::SubsetsImplicitTrieKernel(): allocated(false) {}

template<uint N, bool Proper>
SubsetsImplicitTrieKernel<N, Proper>::~SubsetsImplicitTrieKernel() {
  deallocate();
}

template<uint N, bool Proper>
void SubsetsImplicitTrieKernel<N, Proper>::run(
    uint small_pos,
    uint small_cnt,
    const Subset<N>* large,
    uint large_cnt,
    bool* ret) {
  checkCudaErrors(hipMemcpy(large_ptr, large, sizeof(Subset<N>) * large_cnt, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(results, 0, sizeof(bool) * large_cnt));

  if (Proper) {
    subsets_trie_proper_kernel_impl_v2<N, Subset<N>::buckets()><<<(large_cnt + BLK - 1) / BLK, BLK/*, 0, stream*/>>>(
      small_ptr + small_pos, large_ptr, small_cnt, large_cnt, results);
  } else {
    subsets_trie_kernel_impl_v2<N, Subset<N>::buckets()><<<(large_cnt + BLK - 1) / BLK, BLK/*, 0, stream*/>>>(
      small_ptr + small_pos, large_ptr, small_cnt, large_cnt, results);
  }
  // if (Proper) {
  //   subsets_trie_proper_kernel_impl<N, Subset<N>::buckets()><<<large_cnt, 32/*, 0, stream*/>>>(
  //     small_ptr + small_pos, large_ptr, small_cnt, results);
  // } else {
  //   subsets_trie_kernel_impl<N, Subset<N>::buckets()><<<large_cnt, 32/*, 0, stream*/>>>(
  //     small_ptr + small_pos, large_ptr, small_cnt, results);
  // }
  getLastCudaError("SubsetsImplicitTrieKernel run failed\n"); // TODO: better error message

  checkCudaErrors(hipMemcpy(ret, results, sizeof(bool) * large_cnt, hipMemcpyDeviceToHost));
}

template<uint N, bool Proper>
void SubsetsImplicitTrieKernel<N, Proper>::allocate(const Subset<N>* small, uint small_cnt, uint large_cnt) {
  deallocate();
  allocated = true;

  // checkCudaErrors(hipStreamCreate(&stream));

  checkCudaErrors(hipMalloc((void **) &small_ptr, sizeof(Subset<N>) * small_cnt));
  checkCudaErrors(hipMalloc((void **) &large_ptr, sizeof(Subset<N>) * large_cnt));
  checkCudaErrors(hipMalloc((void **) &results, sizeof(bool) * large_cnt));

  checkCudaErrors(hipMemcpy(small_ptr, small, sizeof(Subset<N>) * small_cnt, hipMemcpyHostToDevice));
}

template<uint N, bool Proper>
void SubsetsImplicitTrieKernel<N, Proper>::deallocate() {
  if (!allocated) {
    return;
  }

  checkCudaErrors(hipFree(small_ptr));
  checkCudaErrors(hipFree(large_ptr));
  checkCudaErrors(hipFree(results));

  // checkCudaErrors(hipStreamDestroy(stream));

  allocated = false;
}

template class SubsetsImplicitTrieKernel<AUT_N, false>;
template class SubsetsImplicitTrieKernel<AUT_N, true>;

} // namespace synchrolib

#undef BLK