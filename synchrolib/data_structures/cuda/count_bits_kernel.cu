#include "hip/hip_runtime.h"
#include "preprocessed_transition_kernel.hpp"

#include <external/cuda/hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <synchrolib/data_structures/subset.hpp>
#include <synchrolib/utils/general.hpp>

#include <chrono>
#include <iostream>

#define BLK 262144

namespace {

template<synchrolib::uint N>
__global__ void count_bits_kernel_impl(const synchrolib::Subset<N>* subsets, int* acc, size_t cnt) {
  int start = blockIdx.x * BLK;
  int end = min(static_cast<size_t>((blockIdx.x + 1) * BLK), cnt);
  int b = threadIdx.x;

  int bucket = b / synchrolib::SUBSETS_BITS;
  int mask = (unsigned long long)1 << (b % synchrolib::SUBSETS_BITS);
  int sum = 0;

  for (int i = start; i < end; ++i) {
    if (subsets[i].v[bucket] & mask) {
      ++sum;
    }
  }

  atomicAdd(acc + b, sum);
}

} // namespace

namespace synchrolib {


template<uint N>
void count_bits_kernel(const Subset<N>* subsets, int* acc, size_t cnt) {
  const Subset<N>* subsets_ptr;
  int* acc_ptr;

std::chrono::steady_clock::time_point begin, end;
begin = std::chrono::steady_clock::now();
  checkCudaErrors(hipMalloc((void **) &subsets_ptr, sizeof(Subset<N>) * cnt));
  checkCudaErrors(hipMalloc((void **) &acc_ptr, sizeof(int) * N));

  checkCudaErrors(hipMemcpy((void*) subsets_ptr, (void*) subsets, sizeof(Subset<N>) * cnt, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(acc_ptr, 0, sizeof(int) * N));
end = std::chrono::steady_clock::now();
std::cout << "Copy time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;

begin = std::chrono::steady_clock::now();
  count_bits_kernel_impl<N><<<(cnt + BLK - 1) / BLK, N/*, 0, stream*/>>>(
    subsets_ptr, acc_ptr, cnt);
  getLastCudaError("count_bits_kernel run failed\n"); // TODO: better error message

  checkCudaErrors(hipMemcpy(acc, acc_ptr, sizeof(int) * N, hipMemcpyDeviceToHost));
end = std::chrono::steady_clock::now();
std::cout << "Gpu time = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;

  checkCudaErrors(hipFree((void*) subsets_ptr));
  checkCudaErrors(hipFree((void*) acc_ptr));
}

template void count_bits_kernel<AUT_N>(const Subset<AUT_N>* subsets, int* acc, size_t cnt);

}  // namespace synchrolib

#undef BLK
